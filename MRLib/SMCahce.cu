#include "hip/hip_runtime.h"
///* MapDSW is a MapReduce Framework which was aimed to fully develop the potential
// * of GPU. It is for an undergraduate graduation thesis at CS/SJTU
// *
// * SMCahce.cu
// *
// *  Created on: 2013-4-23
// *      Author: Shiwei Dong
// */
//
//
//#include "SMCache.h"
//
////insert or update the value in SMCache
//__device__ void SMCache::insert(){
//	//if not overflow
//	//insert into the SMCache
//
//	//if SMCache overflows swap all the gid
//	//merge the cache to the global memory
//	mem_alloc_d.Merge-Cache(cache[gid]);
//	//flush the cache
//	flush();
//	cudasync();
//	//insert into the SMCache;
//
//}
//
//

#include "assert.h"
#include "Common.h"
#include "SMCache.h"
#include "MemAlloc.h"
#include "Intermediate.h"
#include "../UtilLib/hash.h"
#include "../UtilLib/GpuUtil.h"
#include "../UserDef/Mapreduce.h"

__shared__ unsigned int domerge;

__device__ void SMCache::init() {
	for (int i = 0; i < CACHE_POOL; i++) {
		memoryPool[i] = 0;
	}
	for (int j=0; j < CACHE_BUCKETS;j++){
		key_index[j]=0;
		value_index[j]=0;
		key_size[j]=0;
		value_size[j]=0;
		lock[j]=0;	}
	buckets_remain = CACHE_BUCKETS;
	offset = 0;
}

__device__ void SMCache::flush() {
	for (int i = 0; i < CACHE_POOL; i++) {
		memoryPool[i] = 0;
	}
	for (int j=0; j < CACHE_BUCKETS;j++){
		key_index[j]=0;
		value_index[j]=0;
		key_size[j]=0;
		value_size[j]=0;
		lock[j]=0;
	}
	buckets_remain = CACHE_BUCKETS;
	offset = 0;
}

/*Allocate memory from the SMCache memory pool. If success, return the offset. else return -1*/
__device__ int SMCache::Cache_Alloc(unsigned int size) {
	if (buckets_remain > 0 && (offset + size) < CACHE_POOL) {
		unsigned int result = atomicAdd(&offset, size);
		//double check if the offset does not overflow
		if (offset < CACHE_POOL) {
			return result;
		}
		return -1;
	}
	return -1;
}

__device__ void* SMCache::getaddress(unsigned int offset) {
	return memoryPool + offset;
}


//get intermediate from cache buckets which is used while merged into the Mem_Alloc
__device__ bool SMCache::getIntermediate(Intermediate * result, unsigned int bucket){
	assert(bucket<CACHE_BUCKETS);

	if(bucket<CACHE_BUCKETS||key_index[bucket]!=0){
		unsigned short keysize=key_size[bucket];
		unsigned short valuesize=value_size[bucket];
		result->init(getaddress(key_index[bucket]), keysize, getaddress(value_index[bucket]), valuesize);
		return true;
	}

	return false;
}


/**
 * This function perform as a insert and update function in SMCache.
 * The input is the intermediate date which is emitted at the end of the Map function
 */
__device__ void SMCache::insert(Intermediate *inter, MemAlloc* mem_alloc_d) {

	unsigned int tid = threadIdx.x;
	unsigned int Num_threads_b=blockDim.x;
	unsigned int threadsPerGroup = align(Num_threads_b,CACHEGROUP)/CACHEGROUP;

	/**
	 * if the SMCache is not full, operate the insertion or update
	 */
	//there is a global flag "domerge" to judge whether to merge or not
	bool flag = insertOrUpdate(inter);
	if (flag == false) {
		atomicCAS(&domerge, 0, 1);
	}
	__syncthreads();

	/**
	 * else if the SMCache is full, stop all the threads and then swap the SMCache out and merge to the
	 * memory allocator. Then flush the SMCache, and insert again
	 */
	if (domerge) {
		mem_alloc_d->Merge_SMCache(this);
		__syncthreads();
		if (tid % threadsPerGroup == 0) {
			flush();
		}
		if (tid == 0) {
			atomicExch(&domerge, 0);
		}
	}
	__syncthreads();

	//must assert the intermediate key and value larger than the Cache Pool
	if (flag == false) {
		assert(insertOrUpdate(inter));
	}
}

/**
 * insert or update the value, if success return true, else return false
 */
__device__ bool SMCache::insertOrUpdate(Intermediate* inter) {

	//hash the key in order to store the intermediate key value
	unsigned int hash_result = hash((void*) inter->key, inter->keysize);
	unsigned int result_bucket = hash_result % CACHE_BUCKETS;

	bool rehash = false;

	while (buckets_remain>MAX_REMAIN_BUCKETS_C) {

		//if the key's hash bucket does not contain a value, allocate sm memory to it and store the key, value, keysize and value size
		if (key_index[result_bucket] == 0) {

			//attention: should get lock in order to prevent multiple access to the same bucket at the same time
			if (getLock(&lock[result_bucket])) {

				//alloc space for key,value, and store the key in the memory allocated
				//trick put tmp_offset_value first so that tmp_offset_key cannot be 0
			    int tmp_offset_value = Cache_Alloc(inter->valuesize);
				int tmp_offset_key = Cache_Alloc(inter->keysize);

				//if the alloc failed return false
				if (tmp_offset_key < 0 || tmp_offset_value < 0) {
					return false;
				}

				key_index[result_bucket] = tmp_offset_key;
				void* key_adress = getaddress(tmp_offset_key);
				copyVal(key_adress, (void*) inter->key, inter->keysize);

				value_index[result_bucket] = tmp_offset_value;
				void* value_adress = getaddress(tmp_offset_value);
				copyVal(value_adress, (void*) inter->value, inter->valuesize);

				key_size[result_bucket] = inter->keysize;
				value_size[result_bucket] = inter->valuesize;

				assert(releaseLock(&lock[result_bucket]));
				return true;
			}
			rehash = true;

		} else {
			//else when conflict

			//get the key from bucket, aware that every key or value is ended by \0 so that we can get the key or value easily
			unsigned short currentKeysize = key_size[result_bucket];
			if (inter->keysize == currentKeysize) {
				char *currentkey = (char*) getaddress(currentKeysize);
				if (compare(currentkey, inter->key, currentKeysize)) {
					//the current key is exactly the same as the input key, do the reduce step and update the value

				} else {
					//the current key is not the same, then rehash
					rehash = true;
				}
			} else {
				rehash = true;
			}

		}
		if (rehash == true) {
			result_bucket = (hash_result + 1) % CACHE_BUCKETS;
			rehash = false;
		}
	}

	return false;
}


