#include "hip/hip_runtime.h"
/* MapDSW is a MapReduce Framework which was aimed to fully develop the potential
 * of GPU. It is for an undergraduate graduation thesis at CS/SJTU
 *
 * MemAlloc.cu
 *
 *  Created on: 2013-4-15
 *      Author: Shiwei Dong
 */

#include "assert.h"
#include "Common.h"
#include "MemAlloc.h"
#include "SMCache.h"
#include "Intermediate.h"
#include "../UtilLib/hash.h"
#include "../UtilLib/GpuUtil.h"
#include "../UserDef/Mapreduce.h"

//every block has a copy of this shared array. Since global atomic access use too much time, use 8 copies of offsets
//each copy stores the start address for its warp
__shared__ volatile unsigned int global_mem_offset[8];

/**
 * This function serves for initialize purpose. It is invoked when a kernel is launched.
 */
__device__ void MemAlloc::Start_MA_kernal() {

	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	unsigned int blocknum = gridDim.x;
	unsigned int gid = tid / WARP;

	if (tid % WARP == 0) {
		global_mem_offset[gid] = MEM_POOL * (8 * bid + gid) / (blocknum * 8);
	}
}

//Allocate memory from the Memory Allocator memory pool. assumed to be success
__device__ unsigned int MemAlloc::Mem_Alloc(unsigned int size) {

	unsigned int tid = threadIdx.x;
	unsigned int gid = tid / WARP;
	unsigned int tmp_size = align(size, sizeof(unsigned int))
			/ sizeof(unsigned int);

	//Attention: this place may cause overflow, thus Please use smaller jobs
	unsigned int result = atomicAdd((unsigned int *) &global_mem_offset[gid],
			tmp_size);

	return result;
}

__device__ void* MemAlloc::getaddress(unsigned int offset) {
	return memoryPool + offset;
}

__device__ bool MemAlloc::getIntermediate(Intermediate * result,
		unsigned int bucket) {

	assert(bucket<MEM_BUCKETS);
	if (bucket < MEM_BUCKETS && key_index[bucket] != 0) {
		unsigned short keysize = (unsigned short) key_size[bucket];
		unsigned short valuesize = (unsigned short) value_size[bucket];
		//	printf("test %d\n", key_index[bucket]);

		result->init(getaddress(key_index[bucket]), keysize,
				getaddress(value_index[bucket]), valuesize);

		return true;
	}

	return false;
}

/**
 * This function should fully utilize every thread
 * every thread in a group help merge a number of SMCache buckets into the global memory
 * the threads in one cache group deal with one Cache merge
 */
__device__ void MemAlloc::Merge_SMCache(SMCache* Cache, unsigned int groupid) {
	unsigned int tid = threadIdx.x;
	unsigned int num_threads = blockDim.x;
	unsigned int threads_per_group = align(num_threads, CACHEGROUP) / CACHEGROUP;
	unsigned int gid = tid / threads_per_group;

	if (groupid == gid) {
		for (int i = tid % threads_per_group; i < CACHE_BUCKETS; i +=
				threads_per_group) {

			Intermediate result;
			if (Cache->getIntermediate(&result, i)) {
				//printf("merge!!!!\n");
				printf("valus %d\n", *(unsigned int*) result.value);
				insert(&result);
			}
		}
	}
}

__device__ void MemAlloc::insert(Intermediate* inter) {
//It should be asserted that the memory allocator is able to hold all the emitted intermediate and results
//The volume of a job should be determined during the slicing procedure not here
	assert(insertOrUpdate(inter));
	//printf("succes@@@@@\n");
}

__device__ bool MemAlloc::insertOrUpdate(Intermediate* inter) {

//hash the key in order to store the intermediate key value
	unsigned int hash_result = hash((void*) inter->key, inter->keysize);
	unsigned int result_bucket = hash_result % MEM_BUCKETS;
	//printf("hash_result %d\n",hash_result);
	bool rehash = false;
	bool conflict = false;
	int count = 0;

//if can not find a bucket after 1000 rehash, then assumed that the buckets are full
	while (count < 100000) {

		//if the key's hash bucket does not contain a value, allocate mem memory to it and store the key, value, keysize and value size
		if (conflict == false && key_index[result_bucket] == 0) {

			//attention: should get lock in order to prevent multiple access to the same bucket at the same time
			if (getLock(&lock[result_bucket])) {

				//alloc space for key,value, and store the key in the memory allocated
				unsigned int tmp_offset_value = Mem_Alloc(
						(unsigned int) inter->valuesize);
				unsigned int tmp_offset_key = Mem_Alloc(
						(unsigned int) inter->keysize);

				//the allocations of key value offset are assumed to be successful, if overflow, there will be unknown runtime errors
				key_index[result_bucket] = tmp_offset_key;
				void* key_adress = getaddress(tmp_offset_key);
				copyVal(key_adress, (void*) inter->key,
						(unsigned short) inter->keysize);

				value_index[result_bucket] = tmp_offset_value;
				void* value_adress = getaddress(tmp_offset_value);
				copyVal(value_adress, (void*) inter->value,
						(unsigned short) inter->valuesize);

				key_size[result_bucket] = (unsigned int) inter->keysize;
				value_size[result_bucket] = (unsigned int) inter->valuesize;
				used[result_bucket] = 1;

				assert(releaseLock(&lock[result_bucket]));
				//printf("result_bucket : %d!\n", result_bucket);
				return true;
			}
			conflict = true;
		} else {

			if (getLock(&lock[result_bucket])) {
				//	printf("result_bucket : %d!\n", result_bucket);
				conflict = false;

				unsigned int currentKeysize = key_size[result_bucket];
				unsigned int currentKeyindex = key_index[result_bucket];

				if (inter->keysize == currentKeysize) {
					char *currentkey = (char*) getaddress(currentKeyindex);
					if (compare(currentkey, inter->key, currentKeysize)) {
						//the current key is exactly the same as the input key, do the reduce step and update the value
						Intermediate current;
						getIntermediate(&current, result_bucket);
						printf("result_bucket : %d!\n", currentKeyindex);
						reduce(&current, inter);
						assert(releaseLock(&lock[result_bucket]));
						return true;
					} else {
						//the current key is not the same, then rehash
						rehash = true;
					}
				} else {
					rehash = true;
				}
				assert(releaseLock(&lock[result_bucket]));
			}

			if (rehash == true) {
				result_bucket = (result_bucket + 1) % MEM_BUCKETS;
				count++;
				rehash = false;
			}
		}

	}
	printf("count : %d\n", count);
	return false;
}

