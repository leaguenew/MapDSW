#include "hip/hip_runtime.h"
/* MapDSW is a MapReduce Framework which was aimed to fully develop the potential
 * of GPU. It is for an undergraduate graduation thesis at CS/SJTU
 *
 * MemAlloc.cu
 *
 *  Created on: 2013-4-15
 *      Author: Shiwei Dong
 */

#include "assert.h"
#include "Common.h"
#include "MemAlloc.h"
#include "SMCache.h"
#include "Intermediate.h"
#include "../UtilLib/hash.h"
#include "../UtilLib/GpuUtil.h"
#include "../UserDef/Mapreduce.h"

//#include "sm_11_atomic_functions.h"

////the global data in the device memory
//global_data_t* global_data_d;
//
////the offset
//unsigned int* input_offset_d;
//unsigned int* input_size_d;

//every block has a copy of this shared array. Since global atomic access use too much time, use 8 copies of offsets
//each copy stores the start address for its warp
__shared__ volatile unsigned int global_mem_offset[8];

/**
 * This function serves for initialize purpose. It is invoked when a kernel is launched.
 */
__device__ void MemAlloc::Start_MA_kernal() {

	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	unsigned int blocknum = gridDim.x;
	unsigned int gid = tid / WARP;

	if (tid % WARP == 0) {
		global_mem_offset[gid] = MEM_POOL * (8 * bid + gid) / (blocknum * 8);
		//printf("bid %d gid %d offset %d: \n",bid,gid,global_mem_offset[gid]);
		//if(bid==0&&tid==64) printf("global_mem_offset %d \n", global_mem_offset[gid]);
	}
}

/*Allocate memory from the Memory Allocator memory pool. If success, return the offset. else return -1*/
__device__ unsigned int MemAlloc::Mem_Alloc(unsigned int size) {

	unsigned int tid = threadIdx.x;
	unsigned int gid = tid / WARP;
	unsigned int tmp_size=align(size,sizeof(unsigned int))/sizeof(unsigned int);

	//Attention: this place may cause overflow, thus Please use smaller jobs
	unsigned int result = atomicAdd((unsigned int *) &global_mem_offset[gid],
			tmp_size);

	return result;
}

__device__ void* MemAlloc::getaddress(unsigned int offset) {
	return memoryPool + offset;
}

__device__ void MemAlloc::Merge_SMCache(SMCache* Cache, unsigned int groupid) {
//This function should fully utilize every thread
//every thread in a group help merge a number of SMCache buckets into the global memory
//the threads in one cache group deal with one Cache merge

	unsigned int tid = threadIdx.x;
	unsigned int num_threads = blockDim.x;
	unsigned int threads_per_group = align(num_threads, CACHEGROUP) / CACHEGROUP;
	unsigned int gid = tid / threads_per_group;

	if (groupid == gid) {
		for (int i = tid % threads_per_group; i < CACHE_BUCKETS; i +=
				threads_per_group) {
			//printf("threads_per_group %d\n",threads_per_group);
			Intermediate result;
			//	if(tid==160)printf("Group :%d, i %d\n", gid, i);
			//if(gid==7){printf(" i %d tid %d\n",i, tid);}
			if (Cache->getIntermediate(&result, i)) {
				//printf("merging from SM to Device memory %d\n", threadIdx.x);
				//printf("ssssssssgid %d, i %d tid %d\n", gid, i, tid);
			//	if(blockIdx.x==0)printf("inter value result %d\n",(int)*((char*)result.value));
				insert(&result);
			}
		}
	}
}

//It is different from the SMCache insert
__device__ void MemAlloc::insert(Intermediate* inter) {
//It should be asserted that the memory allocator is able to hold all the emitted intermediate and results
//The volume of a job should be determined during the slicing procedure not here
	assert(insertOrUpdate(inter));
}

__device__ bool MemAlloc::insertOrUpdate(Intermediate* inter) {

//hash the key in order to store the intermediate key value
	unsigned int hash_result = hash((void*) inter->key, inter->keysize);
	unsigned int result_bucket = hash_result % MEM_BUCKETS;

	bool rehash = false;
	int count = 0;

//may overflow when all the buckets are full, please avoid such situation
//if can not find a bucket after 1000 rehash, then assumed that the buckets are full
	while (count < 1000) {

		//if the key's hash bucket does not contain a value, allocate sm memory to it and store the key, value, keysize and value size
		if (key_index[result_bucket] == 0) {

			//attention: should get lock in order to prevent multiple access to the same bucket at the same time
			if (getLock(&lock[result_bucket])) {

				//alloc space for key,value, and store the key in the memory allocated
				unsigned int tmp_offset_key = Mem_Alloc(
						(unsigned int) inter->keysize);
				unsigned int tmp_offset_value = Mem_Alloc(
						(unsigned int) inter->valuesize);
			//	if(blockIdx.x==3)printf("tmp_offset_key %d tmp_offset_value %d\n",tmp_offset_key,tmp_offset_value );
				//the allocations of key value offset are assumed to be successful, if overflow, there will be unknown runtime errors
				key_index[result_bucket] = tmp_offset_key;
				void* key_adress = getaddress(tmp_offset_key);
				copyVal(key_adress, (void*) inter->key,
						(unsigned short) inter->keysize);

				value_index[result_bucket] = tmp_offset_value;
				void* value_adress = getaddress(tmp_offset_value);
				copyVal(value_adress, (void*) inter->value,
						(unsigned short) inter->valuesize);
//				printf("inter->value %d value %d\n",*(unsigned int*)inter->value,*(unsigned int*)(char*)getaddress(value_index[result_bucket]) );

				key_size[result_bucket] = (unsigned int) inter->keysize;
				value_size[result_bucket] = (unsigned int) inter->valuesize;
				used[result_bucket] = 1;

				assert(releaseLock(&lock[result_bucket]));
				return true;
			}
			rehash = true;

		} else {
			//else when conflict
			//printf("conflict!!!!\n");
			//get the key from bucket, aware that every key or value is ended by \0 so that we can get the key or value easily		unsigned short currentKeysize = key_size[result_bucket];
//			unsigned int currentKeysize = key_size[result_bucket];
//			unsigned int currentKeyindex = key_index[result_bucket];
//
//			if (inter->keysize == currentKeysize) {
//				char *currentkey = (char*) getaddress(currentKeyindex);
//				if (compare(currentkey, inter->key, currentKeysize)) {
//					//the current key is exactly the same as the input key, do the reduce step and update the value
//					printf("same!!!!!!!\n");
//					reduce();
//					return true;
//				} else {
//					//the current key is not the same, then rehash
//					rehash = true;
//				}
//			} else {
//				rehash = true;
//			}
			rehash=true;

		}

		if (rehash == true) {
			result_bucket = (result_bucket + 1) % MEM_BUCKETS;
			rehash = false;
		}
		count++;
	}
	//printf("count is %d\n", count);
	return false;
}

