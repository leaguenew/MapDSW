#include "hip/hip_runtime.h"
/* MapDSW is a MapReduce Framework which was aimed to fully develop the potential
 * of GPU. It is for an undergraduate graduation thesis at CS/SJTU
 *
 * GpuUtil.cu
 *
 *  Created on: 2013-5-3
 *      Author: Shiwei Dong
 */

#include "GpuUtil.h"
#include "../MRLib/Common.h"

__device__ uint32_t getThreadID() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ uint32_t getNumThreads() {
	return (gridDim.y * gridDim.x) * (blockDim.x * blockDim.y * blockDim.z);
}

__device__ unsigned int align(unsigned int size, unsigned int ALIGN) {

	return (size + ALIGN - 1) & (~(ALIGN - 1));
}

__device__ void copyVal(void *dst, void *src, unsigned short size) {

	char *d = (char*) dst;
	const char *s = (const char *) src;
	for (unsigned short i = 0; i < size; i++)
		d[i] = s[i];
}

//the bucket is locked when lock==1, initially 0
__device__ bool getLock(int* lock) {
	return atomicCAS(lock, 0, 1) == 0;
}

//the bucket is locked when lock==1, initially 0
__device__ bool releaseLock(int* lock) {
	return atomicCAS(lock, 1, 0) == 1;

}

__host__ void memcpyD2H(void * dst, const void * src, unsigned int size) {
	CE(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

__host__ void memcpyH2D(void * dst, const void * src, unsigned int size) {
	CE(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

